// MIT License
//
// Copyright (c) Mvine Ltd. All rights reserved.
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.


#include <hip/hip_runtime.h>

#include <algorithm>
#include <iostream>
#include <vector>

#include <cassert>
#include <cstddef>

// HIP_CHECK copied from Common/example_utils.hpp
constexpr int error_exit_code = -1;

#define CUDA_CHECK(condition)                                                                \
    {                                                                                       \
        const hipError_t error = condition;                                                 \
        if(error != hipSuccess)                                                             \
        {                                                                                   \
            std::cerr << "An error encountered: \"" << hipGetErrorString(error) << "\" at " \
                      << __FILE__ << ':' << __LINE__ << std::endl;                          \
            std::exit(error_exit_code);                                                     \
        }                                                                                   \
    }

constexpr unsigned int BLOCK_SIZE = 16;


__global__ void gpu_transpose_2D_array(float *in, float *transposed, size_t rows, int cols);
   
__global__ void gpu_transpose_2D_array(float *in, float *transposed, size_t rows, size_t cols)
{  
        
   unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
   unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
   
   if (xIndex < cols && yIndex < rows)
   {  
       unsigned int index_in  = xIndex + cols * yIndex;
       unsigned int index_out = yIndex + rows * xIndex;
       transposed[index_out] = in[index_in]; 
   }
}     

__global__
void gpu_linear(float *a, float *b,  float *c, float *r, int m, int n, int k);
      
__global__ void gpu_linear(float *a, float *b,  float *c, float *r, int m, int n, int k)
{     
// linear = weights x activation + bias, so r = a x b + c (using the short param names)
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if( col < k && row < m)
    {
        for(size_t i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        r[row * k + col] = sum + c[row];
    }
}   

__global__
void gpu_matmul(float *a, float *b,  float *r, int m, int n, int k);

__global__ void gpu_matmul(float *a, float *b,  float *r, int m, int n, int k)
{
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if( col < k && row < m)
    {
        for(size_t i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        r[row * k + col] = sum ;
    }
}

__global__
void gpu_partial_matmul(float *a, float *b,  float *r, int m, int n, int k, int max_col);

__global__ void gpu_partial_matmul(float *a, float *b,  float *r, int m, int n, int k, int max_col)
{
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if( col < max_col && row < m)
    {
        for(size_t i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        r[row * max_col + col] = sum ;
    }
}


__device__ int unconverged;

__global__
void gpu_matmul_check_converged(float *a, float *b,  float *r, float epsilon, int m, int n, int k);

__global__ void gpu_matmul_check_converged(float *a, float *b,  float *r, float epsilon,int m, int n, int k)
{
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if( col < k && row < m)
    {
        for(size_t i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        r[row * k + col] = sum ;
        if (row != col && abs(sum) > epsilon) {
           unconverged = 1;
        }
    }
}

__global__ 
void gpu_matrix_sigmoid(float *a, float *b, int m, int n );

__global__ void gpu_matrix_sigmoid(float *a,float *b, int m, int n )
{   
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if( col < n && row < m)
    {
        b[row * n + col] = 1 / ( 1 + exp( -1 * a[row * n + col] ) );
    }
}   
    

__global__ void gpu_mse_cost(float *a, float *t, float *o, int m, int n );

__global__ void gpu_mse_cost(float *a, float *t, float *o, int m, int n ) {

    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if( col < n && row < m)
    {
        o[row * n + col] = powf(a[row * n + col] - t[row * n + col],2)/2;
    }
}

__global__ void gpu_cle_cost(float *a, float *t, float *o, int m, int n );

__global__ void gpu_cle_cost(float *a, float *t, float *o, int m, int n ) {

    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if( col < n && row < m)
    {
        float arg1 = a[row * n + col];
        if (arg1 > 0) {
           arg1 = log(a[row * n + col]);
        } else {
           arg1 = 0;
        }
        float arg2 = 1 - a[row * n + col];
        if (arg2 > 0) {
           arg2 = log(1 - a[row * n + col]);
        } else {
           arg2 = 0;
        }
        o[row * n + col] = -t[row * n + col]*arg1-(1-t[row * n + col])*arg2;
    }
}

__global__ void gpu_mse_cost_derivative(float *a, float *t, float *o, int m, int n );
    
__global__ void gpu_mse_cost_derivative(float *a, float *t, float *o, int m, int n ) {
    
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if( col < n && row < m)
    {
        o[row * n + col] = (a[row * n + col] - t[row * n + col]) * ( a[row * n + col] * ( 1 - a[row * n + col] ) );
    }
}   

__global__ void gpu_cle_cost_derivative(float *a, float *t, float *o, int m, int n );

__global__ void gpu_cle_cost_derivative(float *a, float *t, float *o, int m, int n ) {

    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if( col < n && row < m)
    {
        o[row * n + col] = (a[row * n + col] - t[row * n + col]);
    }
}

__global__ void gpu_add_two_same_size(float *a,float *b, size_t m, size_t n);
    
__global__ void gpu_add_two_same_size(float *a,float *b, size_t m, size_t n) {
    
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if( col < n && row < m)
    {
        a[row * n + col] += b[row * n + col];
    }
}   

__global__
void gpu_weight_derivative(float *a, float *b, float *r, int m, int n, int k);
    
__global__ void gpu_weight_derivative(float *a, float *b, float *r, int m, int n, int k)
{   
// weight_prime = delta x activation + current weight_prime, so r = a x b + r (using the short param names)
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if( col < k && row < m)
    {
        for(size_t i = 0; i < n; i++)
        { 
            sum += a[row * n + i] * b[i * k + col];
        } 
        r[row * k + col] = sum + r[row * k + col];
    }
} 

__global__ void gpu_sigmoid_prime(float *a, float *o, int m, int n );
    
__global__ void gpu_sigmoid_prime(float *a, float *o, int m, int n ) {
    
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if( col < n && row < m)
    {
        o[row * n + col] =  a[row * n + col] * ( 1 - a[row * n + col] ) ;
    }
}   
    
__global__
void gpu_derivative(float *a, float *b,  float *c, float *r, int m, int n, int k);
      
__global__ void gpu_derivative(float *a, float *b,  float *c, float *r, int m, int n, int k)
{        
// linear = weights x activation + bias, so r = a x b + c (using the short param names)
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if( col < k && row < m)
    {
        for(size_t i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        r[row * k + col] = sum * c[row * k + col];
    }
}   

__global__ void gpu_update_weights(float modifier, float decay, float *a,float *b, size_t m, size_t n);
    
__global__ void gpu_update_weights(float modifier, float decay, float *a,float *b, size_t m, size_t n) {
    
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if( col < n && row < m)
    {
        a[row * n + col] = decay * a[row * n + col ] - modifier * b[row * n + col];
    }
}   

__global__ void gpu_update_biases(float modifier, float *a,float *b, size_t m, size_t n);

__global__ void gpu_update_biases(float modifier, float *a,float *b, size_t m, size_t n) {

    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if( col < 1 && row < m)
    {
        for(size_t i = 0; i < n; i++)
        {
            sum += b[row * n + i];
        }
        a[row] -= modifier * sum;
    }
}

__global__ void gpu_calc_means(float *data, float *means, size_t height, size_t width);
    
__global__ void gpu_calc_means(float *data, float *means, size_t height, size_t width) {
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if( col < width  && row < 1) // add up each column, so only 1 row needed
    {
           for (size_t j = 0; j < height; j++) {
              means[col] += data[ j * width + col ] / height;
           } 
    }
}   

__global__ void gpu_calc_stddev(float *data, float *means, float *stddev, size_t height, size_t width);
    
__global__ void gpu_calc_stddev(float *data, float *means, float *stddev, size_t height, size_t width) {
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if( col < width  && row < 1) // add up each column, so only 1 row needed
    {
           for (size_t j = 0; j < height; j++) {
              stddev[col] += powf(data[ j * width + col ] - means[col], 2);
           } 
           stddev[col]  = sqrt( stddev[ col ] / (height - 1) );
    }
}   

__global__ void gpu_assign_z_scores(float *data, float *means, float *stddev, float *z, size_t height, size_t width);
    
__global__ void gpu_assign_z_scores(float *data, float *means, float *stddev, float *z, size_t height, size_t width) {
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if( col < width  && row < height)
    { 
           if (stddev[col] == 0) {
              z[row * width + col ] = 0;
           } else {
              z[row * width + col] = (data[row * width + col] - means[col]) / stddev[col];
              //z[row * width + col] = (data[row * width + col] - means[col]);// / stddev[col];
           }
    } 
}   
   
__global__ void gpu_centre_data(float *data, float *means, float *z, size_t height, size_t width);
    
__global__ void gpu_centre_data(float *data, float *means, float *z, size_t height, size_t width) {
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if( col < width  && row < height)
    { 
              z[row * width + col] = (data[row * width + col] - means[col]);
    } 
}   
   
__global__ void gpu_calc_covariance(float *z, float *cov, size_t height, size_t width);
      
__global__ void gpu_calc_covariance(float *z, float *cov, size_t height, size_t width) {
    size_t row = blockIdx.y * blockDim.y + threadIdx.y; 
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if( col < width  && row < width) 
    {    
           float sum = 0;
           if (col == row) {
              for (size_t i = 0; i < height; i++) {
                 sum += powf(z[i * width + col], 2) / height;
              } 
           } else {
              for (size_t i = 0; i < height; i++) {
                 sum += (z[i * width + col] * z[i * width + row]) / height;
              }
           }
           cov[ row * width + col] = sum ;
    }
}  

__global__ void gpu_qr_column_mult(float *orig, float *r, float *dotp, size_t height, size_t width, int this_column);

__global__ void gpu_qr_column_mult(float *orig, float *r, float *dotp, size_t height, size_t width, int this_column) {

    size_t row = blockIdx.y * blockDim.y + threadIdx.y;  
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if( col < this_column && row < 1)
    {
           float dotprod = 0;
           for (size_t j = 0; j < height; j++) {
               dotprod += orig[ j * width + this_column ] * r[j * width + col];
           } 
           dotp[ col ] = dotprod;
    }
}   

__global__ void gpu_qr_column(float *orig, float *r, float *dotp, size_t height, size_t width, int this_column);
    
__global__ void gpu_qr_column(float *orig, float *r, float *dotp, size_t height, size_t width, int this_column) {

    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if( col == this_column && row < height)
    {
        float ax = orig[ row * width + this_column]; 
        r[ row * width + this_column ] = ax;
        for(size_t i = 0; i < this_column; i++)
        {
           float dotprod = dotp[ i ]; 
           //for (size_t j = 0; j < height; j++) {
            //   dotprod += r[ j * width + this_column ] * r[j * width + i];
           //}
           r[ row * width + this_column ] -= r[row * width + i] * dotprod;
        }
    }   
}

__global__ void gpu_qr_l2_norm(float *r, float *l2norm, size_t height, size_t width, int this_column);
    
__global__ void gpu_qr_l2_norm(float *r, float *l2norm, size_t height, size_t width, int this_column) {
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if( col == this_column && row < 1)
    {
        float l2norm = 0;
        for (int i = 0; i < height; i++) {
           l2norm += powf(r[i * width + this_column] , 2);
        }
        l2norm = sqrt(l2norm);
        for (int i = 0; i < height; i++) {
           r[i * width + this_column] /= l2norm; 
        }  
    }
}   

__global__ void gpu_qr_clamp_r_to_0(float *r, size_t height, size_t width);
    
__global__ void gpu_qr_clamp_r_to_0(float *r, size_t height, size_t width) {
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if( col < width  && row < height)
    {
           if (col < row) {
              r[row * width + col] = 0;
           }
    }
}   

__global__ void gpu_eigenvector_signs(float *eigenvectors, float *sums, size_t rows, size_t cols);
    
__global__ void gpu_eigenvector_signs(float *eigenvectors, float *sums, size_t rows, size_t cols) {
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if( col < cols  && row < 1) // one calc per vector
    {
        sums[col] = 0;
        for (int i = 0; i < rows; i++) {
              sums[col] += eigenvectors[i * cols + col];
        }
        if (sums[col] < 1) {
           sums[col] = 0;
           for (int i = 0; i < rows; i++) {
              eigenvectors[i * cols + col] *= -1;
              sums[col] += eigenvectors[i * cols + col];
           }
        }
    }
}   

__global__ void gpu_reorder_eigenvectors(float *unsorted, float *sorted, int *indicies, size_t rows, size_t cols);

__global__ void gpu_reorder_eigenvectors(float *unsorted, float *sorted, int *indicies, size_t rows, size_t cols) {
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if( col < cols  && row < rows) 
    {
       sorted[row * cols + col] = unsorted[row * cols + indicies[col]];
    }
}  

int run_gpu_linear( float *activation, float *device_Weights,  float *device_Bias, float *device_Output, int m, int n, int k) ;
int run_gpu_linear( float *activation, float *device_Weights,  float *device_Bias, float *device_Output, int m, int n, int k) {
    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGridT(grid_cols, grid_rows);
    dim3 dimBlockT(BLOCK_SIZE, BLOCK_SIZE);
    gpu_linear<<<dimGridT, dimBlockT>>>(device_Weights, activation, device_Bias, device_Output,m, n, k);
    return 1;
}  

int run_gpu_matmul( float *a, float *b,  float *r, int input_size, int middle_size, int output_size) ;
int run_gpu_matmul( float *a, float *b,  float *r, int input_size, int middle_size, int output_size) {
    unsigned int grid_rows = (input_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (output_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGridT(grid_cols, grid_rows);
    dim3 dimBlockT(BLOCK_SIZE, BLOCK_SIZE);
    gpu_matmul<<<dimGridT, dimBlockT>>>(a, b, r, input_size,middle_size, output_size);
    return 1;
}

int run_gpu_partial_matmul( float *a, float *b,  float *r, int input_size, int middle_size, int output_size, int max_columns) ;
int run_gpu_partial_matmul( float *a, float *b,  float *r, int input_size, int middle_size, int output_size, int max_columns) {
    unsigned int grid_rows = (input_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (output_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGridT(grid_cols, grid_rows);
    dim3 dimBlockT(BLOCK_SIZE, BLOCK_SIZE);
    gpu_partial_matmul<<<dimGridT, dimBlockT>>>(a, b, r, input_size,middle_size, output_size, max_columns); // only the first "max_columns" of the second matrix are acutally used in the matmul, even though the second matrix might be much larger
    return 1;
}   

int run_gpu_matmul_check_converged( float *a, float *b,  float *r, float epsilon, int input_size, int middle_size, int output_size) ;
int run_gpu_matmul_check_converged( float *a, float *b,  float *r, float epsilon, int input_size, int middle_size, int output_size) {
    unsigned int grid_rows = (input_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (output_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGridT(grid_cols, grid_rows);
    dim3 dimBlockT(BLOCK_SIZE, BLOCK_SIZE);
    gpu_matmul_check_converged<<<dimGridT, dimBlockT>>>(a, b, r, epsilon, input_size,middle_size, output_size);
    return 1;
}

int run_gpu_derivative( float *a, float *b,  float *sp, float *r, int input_size, int middle_size, int output_size) ;
int run_gpu_derivative( float *a, float *b,  float *sp, float *r, int input_size, int middle_size, int output_size) {
    unsigned int grid_rows = (input_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (output_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGridT(grid_cols, grid_rows);
    dim3 dimBlockT(BLOCK_SIZE, BLOCK_SIZE);
    gpu_derivative<<<dimGridT, dimBlockT>>>(a, b, sp, r,input_size, middle_size, output_size);
    return 1;
}   

int run_gpu_weight_derivative( float *a, float *b, float *r, int input_size, int middle_size, int output_size);
int run_gpu_weight_derivative( float *a, float *b, float *r, int input_size, int middle_size, int output_size) {
    unsigned int grid_rows = (input_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (output_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGridT(grid_cols, grid_rows);
    dim3 dimBlockT(BLOCK_SIZE, BLOCK_SIZE);
    gpu_weight_derivative<<<dimGridT, dimBlockT>>>(a, b, r,input_size, middle_size, output_size);
    return 1;
}   

int gpu_sigmoid( float *device_Output, float *device_Activated_Output, int output_size , int cols);
int gpu_sigmoid( float *device_Output, float *device_Activated_Output, int output_size , int cols) {
    unsigned int grid_rows = (output_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE; 
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); 
    
    gpu_matrix_sigmoid<<<dimGrid, dimBlock>>>(device_Output, device_Activated_Output, output_size, cols);
    return 1;
}       

int run_gpu_sigmoid_prime( float *device_Activated_Output, float *device_Activated_Output_Derivative, int output_size, int cols );
int run_gpu_sigmoid_prime( float *device_Activated_Output, float *device_Activated_Output_Derivative, int output_size, int cols ) {
    unsigned int grid_rows = (output_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    gpu_sigmoid_prime<<<dimGrid, dimBlock>>>(device_Activated_Output, device_Activated_Output_Derivative, output_size, cols);
    return 1;
}   

int gpu_add_same_size( float *lhs, float *rhs, int arraysize ); 
int gpu_add_same_size( float *lhs, float *rhs, int arraysize ) {
    unsigned int grid_rows = (arraysize + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (1 + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    gpu_add_two_same_size<<<dimGrid, dimBlock>>>(lhs, rhs, arraysize, 1);
    return 1;
}   

void run_gpu_transpose_2D_array( float *in, float *transpose, size_t rows, size_t cols) ;
void run_gpu_transpose_2D_array( float *in, float *transpose, size_t rows, size_t cols) {
    unsigned int grid_rows = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    gpu_transpose_2D_array<<<dimGrid, dimBlock>>>(in, transpose, rows, cols);
}   

void run_gpu_update_weights( float modifier, float decay, float *device_Weights, float *device_Weights_Derivative, int output_size, int input_size) ;
void run_gpu_update_weights( float modifier, float decay, float *device_Weights, float *device_Weights_Derivative, int output_size, int input_size) {
    unsigned int grid_rows = (output_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (input_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); 
    gpu_update_weights<<<dimGrid, dimBlock>>>(modifier, decay, device_Weights, device_Weights_Derivative, output_size, input_size);
}   

void run_gpu_update_biases( float modifier, float *device_Bias, float *device_Bias_Derivative, int output_size, int batch_size);

void run_gpu_update_biases( float modifier, float *device_Bias, float *device_Bias_Derivative, int output_size, int batch_size){
    unsigned int grid_rows = (output_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (batch_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); 
    gpu_update_biases<<<dimGrid, dimBlock>>>(modifier, device_Bias, device_Bias_Derivative, output_size, batch_size);
}   
    

void gpu_calculate_cost_and_derivative(float *activated_output, float *y, float *cost_derivative, size_t rows, size_t cols, int loss_function) ;
void gpu_calculate_cost_and_derivative(float *activated_output, float *y, float *cost_derivative, size_t rows, size_t cols, int loss_function) { 
    // (output - target) * output_derivative : output_derivative can be derived from output, so do it all within the GPU
    unsigned int grid_rows = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE; 
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    if (loss_function == 2) {
       gpu_cle_cost_derivative<<<dimGrid, dimBlock>>>( activated_output, y, cost_derivative, rows, cols );
    } else {
       gpu_mse_cost_derivative<<<dimGrid, dimBlock>>>( activated_output, y, cost_derivative, rows , cols );
    }
}

void gpu_calculate_cost(float *device_Activated_Output, float *device_y, float *device_Cost, size_t rows, size_t cols, int loss_function);
void gpu_calculate_cost(float *device_Activated_Output, float *device_y, float *device_Cost, size_t rows, size_t cols, int loss_function){
    // (output - target) * output_derivative : output_derivative can be derived from output, so do it all within the GPU
    unsigned int grid_rows = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    
    if (loss_function == 2) {
       gpu_cle_cost<<<dimGrid, dimBlock>>>( device_Activated_Output, device_y, device_Cost, rows, cols );
    } else {
       gpu_mse_cost<<<dimGrid, dimBlock>>>( device_Activated_Output, device_y, device_Cost, rows, cols );
    } 
}      

void run_gpu_calc_means( float *data, float *means, size_t rows, size_t cols ) {
    unsigned int grid_rows = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    gpu_calc_means<<<dimGrid, dimBlock>>>(data, means, rows, cols);
}  

void run_gpu_calc_stddev( float *data, float *means, float *stddev, size_t rows, size_t cols ) {
    unsigned int grid_rows = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); 
    gpu_calc_stddev<<<dimGrid, dimBlock>>>(data, means, stddev, rows, cols);
}   

void run_gpu_assign_z_scores( float *data, float *means, float *stddev, float *z, size_t rows, size_t cols ) {
    unsigned int grid_rows = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    gpu_assign_z_scores<<<dimGrid, dimBlock>>>(data, means, stddev, z, rows, cols);
}
    
void run_gpu_centre_data( float *data, float *means, float *z, size_t rows, size_t cols ) {
    unsigned int grid_rows = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    gpu_centre_data<<<dimGrid, dimBlock>>>(data, means, z, rows, cols);
}
    
void run_gpu_calc_covariance( float *z, float *cov, size_t rows, size_t cols ) {
    unsigned int grid_rows = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    gpu_calc_covariance<<<dimGrid, dimBlock>>>(z, cov, rows, cols);
}             
           
void run_gpu_qr_column_mult( float *orig, float *r, float *dotp, size_t rows, size_t cols, int colno ) {
    unsigned int grid_rows = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    gpu_qr_column_mult<<<dimGrid, dimBlock>>>(orig, r, dotp, rows, cols, colno);
}  

void run_gpu_qr_column( float *orig, float *r, float *dotp, size_t rows, size_t cols, int colno ) {
    unsigned int grid_rows = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows); 
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    gpu_qr_column<<<dimGrid, dimBlock>>>(orig, r, dotp, rows, cols, colno);
}

void run_gpu_qr_l2_norm( float *r, float *l2norm, size_t rows, size_t cols, int colno ) {
    unsigned int grid_rows = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    gpu_qr_l2_norm<<<dimGrid, dimBlock>>>(r, l2norm, rows, cols, colno);
}   

void run_gpu_qr_clamp_r_to_0( float *r, size_t rows, size_t cols ) {
    unsigned int grid_rows = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    gpu_qr_clamp_r_to_0<<<dimGrid, dimBlock>>>(r, rows, cols);
}          

void run_gpu_eigenvector_signs(float *eigenvectors, float *sums, size_t rows, size_t cols);
void run_gpu_eigenvector_signs(float *eigenvectors, float *sums, size_t rows, size_t cols) {
    unsigned int grid_rows = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    gpu_eigenvector_signs<<<dimGrid, dimBlock>>>(eigenvectors, sums, rows, cols);
}

void run_gpu_reorder_eigenvectors(float *unsorted, float *sorted, int *indicies, size_t rows, size_t cols);
void run_gpu_reorder_eigenvectors(float *unsorted, float *sorted, int *indicies, size_t rows, size_t cols) {
    unsigned int grid_rows = (rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    gpu_reorder_eigenvectors<<<dimGrid, dimBlock>>>(unsorted, sorted, indicies, rows, cols);
}


void gpu_memcpy_to_device( float *host_data, float *device_data, size_t size_data);
void gpu_memcpy_to_device( float *host_data, float *device_data, size_t size_data) {
       CUDA_CHECK(hipMemcpy(device_data, host_data, size_data, hipMemcpyHostToDevice));
}

void gpu_memcpy_to_device_int( int *host_data, int *device_data, size_t size_data);
void gpu_memcpy_to_device_int( int *host_data, int *device_data, size_t size_data) {
       CUDA_CHECK(hipMemcpy(device_data, host_data, size_data, hipMemcpyHostToDevice));
}

void gpu_memcpy_from_device( float *host_data, float *device_data, size_t size_data);
void gpu_memcpy_from_device( float *host_data, float *device_data, size_t size_data) {
       CUDA_CHECK(hipMemcpy(host_data, device_data, size_data, hipMemcpyDeviceToHost));
}

void gpu_memcpy_intra_device( float *from_data, float *to_data, size_t size_data);
void gpu_memcpy_intra_device( float *from_data, float *to_data, size_t size_data) {
       CUDA_CHECK(hipMemcpy(to_data, from_data, size_data, hipMemcpyDeviceToDevice));
}

float * gpu_device_malloc( size_t size_data);
float * gpu_device_malloc( size_t size_data) {
       float * device_data;
       CUDA_CHECK(hipMalloc((void**)&device_data, size_data));
       return device_data;
}

int * gpu_device_malloc_int( size_t size_data);
int * gpu_device_malloc_int( size_t size_data) {
       int * device_data;
       CUDA_CHECK(hipMalloc((void**)&device_data, size_data));
       return device_data;
}

float * gpu_host_malloc( size_t size_data);
float * gpu_host_malloc( size_t size_data) {
   float *host_data;
   CUDA_CHECK(hipHostMalloc((void **)&host_data, size_data , hipHostMallocDefault));
   return host_data;
}

void gpu_free_device(void *device_data) {
   CUDA_CHECK(hipFree((void *)device_data));
}
void gpu_free_host(void *host_data) {
   CUDA_CHECK(hipHostFree((void *)host_data));
}

void gpu_reset_unconverged() {
   int some_int = 0;
   CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(unconverged), &some_int, sizeof(int), 0, hipMemcpyHostToDevice));
}
int gpu_get_unconverged_state() {
   int some_int;
   CUDA_CHECK(hipMemcpyFromSymbol(&some_int, HIP_SYMBOL(unconverged), sizeof(int), 0, hipMemcpyDeviceToHost));
   return some_int;
}
